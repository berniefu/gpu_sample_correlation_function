#include "hip/hip_runtime.h"
#define MIN 0
#define MAX 80
#define INTERVAL_NUM 40  // This three parameters determine the histogram range
#define SAMPLE_NUM 30
#define OBJECT_NUM 4096  // Number of objects to sample
#define TOTAL_NUM 1000000  // Total number of objects in the dataset
#define THREAD_LENGTH 16

#include <iostream>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <sstream>

/* kernel routine starts with keyword __global__ */

__global__ void vecadd(double* data_x, double* data_y, double* data_z, int* result, int* final)
{
  int index = blockIdx.x * THREAD_LENGTH * THREAD_LENGTH + threadIdx.x * THREAD_LENGTH + threadIdx.y;
  // initializing
  for (int i = index * INTERVAL_NUM; i < index * INTERVAL_NUM + INTERVAL_NUM; ++i) {
    result[i] = 0;
  }

  double x = data_x[index];
  double y = data_y[index];
  double z = data_z[index];

  // calculation
  for (int i = 0; i < OBJECT_NUM; i++) {
    double distSqr = (data_x[i] - x) * (data_x[i] - x) +
                    (data_y[i] - y) * (data_y[i] - y) +
                    (data_z[i] - z) * (data_z[i] - z);
    int bin_num = (int)((sqrt(distSqr) - MIN - 0.00000001) * INTERVAL_NUM / (MAX - MIN));
    if (bin_num < INTERVAL_NUM)
      result[index * INTERVAL_NUM + bin_num]++;
  }
  
  // move to final result
  for (int i = 0; i < INTERVAL_NUM; ++i) {
    atomicAdd(final + i, result[index * INTERVAL_NUM + i]);
  }
}

using namespace std;

void generateData(int num_point, double box_size) {
  ofstream out("data_file", ios::out);
  for (int i = 0; i < num_point; ++i) {
    for (int j = 0; j < 3; j++) {
      out << (double)rand() / RAND_MAX * box_size << " ";
    }
    out << endl;
  }
  out.close();
}

int main(int argc, char * argv[])
{
  // generateData(1000000, 40.0);  // Use this to generate random data

  clock_t start = clock();
  srand(time(NULL));

  double*** data; // SAMPLE_NUM, OBJECT_NUM, dimensions(3)
  double** result; // SAMPLE_NUM, INTERVAL_NUM
  
  data = new double**[SAMPLE_NUM];
  for (int i = 0; i < SAMPLE_NUM; ++i) {
    data[i] = new double*[OBJECT_NUM];
    for (int j = 0; j < OBJECT_NUM; ++j) {
      data[i][j] = new double[3];
    }
  }
  
  result = new double*[SAMPLE_NUM];
  for (int i = 0; i < SAMPLE_NUM; ++i) {
    result[i] = new double[INTERVAL_NUM];
    for (int j = 0; j < INTERVAL_NUM; ++j) {
      result[i][j] = 0;
    }
  }

  // input data
  ifstream in("data_file", ios::in);
  double x, y, z;
  int num_object = 0;
  while ((in >> x) != NULL) {
    in >> y;
    in >> z;
    for (int i = 0; i < SAMPLE_NUM; ++i) {
      bool replace = false;
      int replace_index = -1;
      if (num_object < OBJECT_NUM) {
        replace = true;
        replace_index = num_object;
      } else {
        int draw = (int) (floor((double)rand() / RAND_MAX * (num_object + 1)));
        if (draw < OBJECT_NUM) {
          replace = true;
          replace_index = draw;
        }
      }
      if (replace) {
        data[i][replace_index][0] = x;
        data[i][replace_index][1] = y;
        data[i][replace_index][2] = z;
      }
    }
    num_object++;
  }

  cout << "After sampling: " << ((double)clock() - start) / CLOCKS_PER_SEC << endl;

  double *host_x, *host_y, *host_z;
  int *host_final;
  double *dev_x, *dev_y, *dev_z;
  int *dev_result, *dev_final;

  /* 1. allocate host memory */
  host_x = (double*)malloc( OBJECT_NUM*sizeof(double));
  host_y = (double*)malloc( OBJECT_NUM*sizeof(double) );
  host_z = (double*)malloc( OBJECT_NUM*sizeof(double) );
  host_final = (int*)malloc( INTERVAL_NUM*sizeof(int) );

  /* 2. allocate GPU memory */
  hipMalloc( &dev_x, OBJECT_NUM*sizeof(double) );
  hipMalloc( &dev_y, OBJECT_NUM*sizeof(double) ); 
  hipMalloc( &dev_z, OBJECT_NUM*sizeof(double) ); 
  hipMalloc( &dev_result, OBJECT_NUM*INTERVAL_NUM*sizeof(int) );
  hipMalloc( &dev_final, INTERVAL_NUM*sizeof(int) );

  for (int s = 0; s < SAMPLE_NUM; ++s){
    for (int i = 0; i < INTERVAL_NUM; ++i)
      host_final[i] = 0;

    /* initialize input data */
    for (int i = 0 ; i < OBJECT_NUM ; i++) {
      host_x[i] = data[s][i][0];
      host_y[i] = data[s][i][1];
      host_z[i] = data[s][i][2];
    }

    /* 3. Copydata (host_x, host_y and host_z) to GPU */
    hipMemcpy( dev_x, host_x, OBJECT_NUM*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_y, host_y, OBJECT_NUM*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_z, host_z, OBJECT_NUM*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( dev_final, host_final, INTERVAL_NUM*sizeof(int), hipMemcpyHostToDevice );

    /* 4. call kernel routine to execute on GPU */
    /* launch 1 thread per vector-element, 1024 threads per block */
    dim3 threadsPerBlock(THREAD_LENGTH, THREAD_LENGTH);
    int numBlocks(OBJECT_NUM / threadsPerBlock.x / threadsPerBlock.y);
    vecadd<<<numBlocks, threadsPerBlock>>>( dev_x, dev_y, dev_z, dev_result, dev_final);

    cout << "After " << s << " samples: " << ((double)clock() - start) / CLOCKS_PER_SEC << endl;

    //!!!!! not n! but interval_num. gpu is wierd though...
    /* transfer results from GPU to CPU */
    hipMemcpy( host_final, dev_final, INTERVAL_NUM*sizeof(int), hipMemcpyDeviceToHost );
 
    for (int i = 0; i < INTERVAL_NUM; i++)
      result[s][i] = (host_final[i] / 2);  // Every pair is counted twice.
  }

  /* free host and GPU memory */
  free(host_x);  
  free(host_y);
  free(host_z);
  free(host_final);
  hipFree(dev_x);
  hipFree(dev_y);
  hipFree(dev_z);
  hipFree(dev_result);
  hipFree(dev_final);
 
  for (int i = 0; i < INTERVAL_NUM; ++i) {
    double Ex = 0, Ex2 = 0;
    for (int j = 0; j < SAMPLE_NUM; ++j) {
      double reg = result[j][i] * TOTAL_NUM * TOTAL_NUM / OBJECT_NUM / OBJECT_NUM;
      Ex += reg;
      Ex2+= reg * reg;
    }
    Ex /= SAMPLE_NUM;
    Ex2/= SAMPLE_NUM;
    double var = Ex2 - Ex * Ex;
    double std = sqrt(var);
    double err95 = (Ex > 0.000001)? std / sqrt(SAMPLE_NUM - 1) * 2 / Ex : 0;
    cout << Ex << " " << err95 << endl;  // Output mean, and relative error for 95% confidence interval
  }

  cout << "End: " << ((double)clock() - start) / CLOCKS_PER_SEC << endl;

  return( 0 );
}
